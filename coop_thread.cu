
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 512

__global__ void add(int *a, int *b);

int main()
{
	int *a, *b;
	int *d_a, *d_b;
	int i;

	// allocate space for device copies
	hipMalloc(&d_a, N*sizeof(int));
	hipMalloc(&d_b, N*sizeof(int));
	//cudaMalloc(&d_c, sizeof(int));

	// allocate variables
	a = (int *)malloc(N*sizeof(int));
	b = (int *)malloc(N*sizeof(int));

	// attribute values to arrays
	for(i = 0; i < N; i++)
		a[i] = i;
	b[0] = 0;	

	// copy inputs to device
	hipMemcpy(d_a, a, N*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, N*sizeof(int), hipMemcpyHostToDevice);

	// Lauch add() kernel on GPU
	add<<<1,N>>>(d_a, d_b);

	// copy result back to Host
	hipMemcpy(b, d_b, N*sizeof(int), hipMemcpyDeviceToHost);

	printf("result = %d\n", b[0]);

	free(a);
	free(b);
	hipFree(d_a);
	hipFree(d_b);
}

__global__ void add(int *a, int *b)
{
	__shared__ int data[N];
	int i;

	// each thread loads one element from global to shared mem
	int index = threadIdx.x + blockIdx.x*blockDim.x;
	data[threadIdx.x] = a[index];
	__syncthreads();

	// do reduction in shared mem
	for(i = 1; i < blockDim.x; i = i*2)
	{
		index = 2*i*threadIdx.x;
		if(index < blockDim.x)
			data[index] = data[index] + data[index + i];
		__syncthreads();
	}

	// write result for this block to global mem
	if(threadIdx.x == 0)
		b[0] = data[0];
}
