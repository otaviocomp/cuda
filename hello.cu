
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>

__global__ void add(int *a, int *b, int *c);

int main()
{
	clock_t t;
	int a, b, c;
	int *d_a, *d_b, *d_c;

	t = clock();
	// allocate space for device copies
	hipMalloc(&d_a, sizeof(int));
	hipMalloc(&d_b, sizeof(int));
	hipMalloc(&d_c, sizeof(int));

	// setup inputs
	a = 1;
	b = 2;

	// copy inputs to device
	hipMemcpy(d_a, &a, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_b, &b, sizeof(int), hipMemcpyHostToDevice);

	// Lauch add() kernel on GPU
	add<<<1,3>>>(d_a, d_b, d_c);

	// copy result back to Host
	hipMemcpy(&c, d_c, sizeof(int), hipMemcpyDeviceToHost);
	t = clock() - t;

	printf("result = %d\n time = %e\n", c, (double)t/CLOCKS_PER_SEC);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
}

__global__ void add(int *a, int *b, int *c)
{
	*c = *a + *b;
}
