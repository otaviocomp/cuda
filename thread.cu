
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 256

__global__ void add(int *a, int *b, int *c);

int main()
{
	int *a, *b, *c;
	int *d_a, *d_b, *d_c;
	int i;

	// allocate space for device copies
	hipMalloc(&d_a, N*sizeof(int));
	hipMalloc(&d_b, N*sizeof(int));
	hipMalloc(&d_c, N*sizeof(int));

	// allocate variables
	a = (int *)malloc(N*sizeof(int));
	b = (int *)malloc(N*sizeof(int));
	c = (int *)malloc(N*sizeof(int));

	// attribute values to arrays
	for(i = 0; i < N; i++)
	{
		a[i] = i;
		b[i] = i;
	}

	// copy inputs to device
	hipMemcpy(d_a, a, N*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, N*sizeof(int), hipMemcpyHostToDevice);

	// Lauch add() kernel on GPU
	add<<<1,N>>>(d_a, d_b, d_c);

	// copy result back to Host
	hipMemcpy(c, d_c, N*sizeof(int), hipMemcpyDeviceToHost);

	for(i = 0; i < N; i++)
		printf("c[%d] = %d\n", i + 1, c[i]);

	free(a);
	free(b);
	free(c);
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
}

__global__ void add(int *a, int *b, int *c)
{
	c[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];
}
